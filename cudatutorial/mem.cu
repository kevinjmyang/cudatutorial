#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <windows.h>

#define MEGABYTE    (1024 * 1024)
#define CHECK(call)                                                          \
{                                                                            \
    const hipError_t error = call;                                          \
    if (error != hipSuccess)                                                \
    {                                                                        \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                        \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                             \
    }                                                                        \
}
double seconds() {
    return (double)clock() / CLOCKS_PER_SEC;
}

int main(int argc, char** argv)
{
    int dev = 0;
    CHECK(hipSetDevice(dev));

    if (argc != 2) {
        printf("usage: %s <size-in-mbs>\n", argv[0]);
        return 1;
    }

    int n_mbs = atoi(argv[1]);
    unsigned int nbytes = n_mbs * MEGABYTE;

    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting at ", argv[0]);
    printf("device %d: %s nbyte %5.2fMB\n", dev,
        deviceProp.name, nbytes / (1024.0f * 1024.0f));

    // allocate the host memory
    double start = seconds();
    float* h_a = (float*)malloc(nbytes);
    double elapsed = seconds() - start;
    printf("Host memory allocation took %2.10f us\n", elapsed * 1000000.0);

    // allocate the device memory
    float* d_a;
    CHECK(hipMalloc((float**)&d_a, nbytes));

    // initialize the host memory
    for (unsigned int i = 0; i < nbytes / sizeof(float); i++) h_a[i] = 0.5f;

    // transfer data from the host to the device
    CHECK(hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice));

    // transfer data from the device to the host
    CHECK(hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost));

    // free memory
    CHECK(hipFree(d_a));

    LARGE_INTEGER frequency;
    LARGE_INTEGER StartingTime, EndingTime;
    double ElapsedTime;
    QueryPerformanceFrequency(&frequency);
    QueryPerformanceCounter(&StartingTime);

    free(h_a);
    QueryPerformanceCounter(&EndingTime);
    ElapsedTime = (double)(EndingTime.QuadPart - StartingTime.QuadPart) / frequency.QuadPart;
    printf("Elapsed Time: %f seconds", ElapsedTime);
    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
